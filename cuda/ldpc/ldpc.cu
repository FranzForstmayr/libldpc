#include "hip/hip_runtime.h"
#include "ldpc.cuh"
#include <exception>
#include <math.h>

using namespace std;
using namespace ldpc;


void* Cuda_Mgd_cl::operator new(size_t len)
{
	void* ptr;
	hipMallocManaged(&ptr, len);
	hipDeviceSynchronize();
	return ptr;
}

void Cuda_Mgd_cl::operator delete(void* ptr)
{
	hipDeviceSynchronize();
	hipFree(ptr);
}


Ldpc_Code_cl::Ldpc_Code_cl(const char* filename, const char* clfile, const bool& mgd)
{
	if (mgd)
	{
		setup_code_mgd(filename);
		setup_layers_mgd(clfile);
		prefetch_code();
	}
	else
	{
		setup_code(filename);
		setup_layers(clfile);
	}

	isMgd = mgd;
}


Ldpc_Code_cl::~Ldpc_Code_cl()
{
	if (isMgd)
	{
		destroy_code_mgd();
	}
	else
	{
		destroy_code();
	}
}


void Ldpc_Code_cl::setup_code(const char* filename)
{
	//init
	puncture_c = nullptr;
	shorten_c = nullptr;
	cw_c = nullptr;
	cn_c = nullptr;
	vw_c = nullptr;
	vn_c = nullptr;
	r_c = nullptr;
	c_c = nullptr;
	lw_c = nullptr;
	layers_c = nullptr;

	try
	{
		FILE *fp;

		fp = fopen(filename, "r");
		if(!fp)
			throw runtime_error("can not open codefile for reading.");

		fscanf(fp, "nc: %lu\n", &n_c);
		fscanf(fp, "mc: %lu\n", &m_c);
		fscanf(fp, "nct: %lu\n", &nct_c);
		fscanf(fp, "mct: %lu\n", &mct_c);
		fscanf(fp,  "nnz: %lu\n", &nnz_c);
		k_c = n_c-m_c;
		kct_c = nct_c-mct_c;

		fscanf(fp, "puncture [%lu]: ", &(num_puncture_c));
		num_puncture_sys_c = 0;
		num_puncture_par_c = 0;
		if(num_puncture_c != 0)
		{
			puncture_c = new size_t[num_puncture_c];
			for(size_t i = 0; i < num_puncture_c; i++)
			{
				fscanf(fp, " %lu ", &(puncture_c[i]));
				if(puncture_c[i] < k_c)
					num_puncture_sys_c++;
				else
					num_puncture_par_c++;
			}
		}

		fscanf(fp, "shorten [%lu]: ", &num_shorten_c);
		if(num_shorten_c != 0)
		{
			shorten_c = new size_t[num_shorten_c];
			for(size_t i = 0; i < num_shorten_c; i++)
				fscanf(fp, " %lu ", &(shorten_c[i]));
		}


		size_t* cw_tmp;
		size_t* vw_tmp;
		cw_c = new uint64_t[m_c] ();
		cw_tmp = new uint64_t[m_c] ();
		vw_c = new uint64_t[n_c] ();
		vw_tmp = new uint64_t[n_c] ();
		r_c = new uint64_t[nnz_c] ();
		c_c = new uint64_t[nnz_c] ();


		for(size_t i = 0; i < nnz_c; i++)
		{
			fscanf(fp, "%lu %lu\n", &(r_c[i]), &(c_c[i]));
			cw_c[r_c[i]]++;
			vw_c[c_c[i]]++;
		}

		cn_c = new size_t*[m_c] ();
		for(size_t i = 0; i < m_c; i++)
			cn_c[i] = new size_t[cw_c[i]] ();

		vn_c = new size_t*[n_c] ();
		for(size_t i = 0; i < n_c; i++)
			vn_c[i] = new size_t[vw_c[i]] ();

		for(size_t i = 0; i < nnz_c; i++)
		{
			cn_c[r_c[i]][cw_tmp[r_c[i]]++] = i;
			vn_c[c_c[i]][vw_tmp[c_c[i]]++] = i;
		}

		delete[] cw_tmp;
		delete[] vw_tmp;

		max_dc_c = 0;
		for(size_t i = 0; i < m_c; i++)
		{
			if(cw_c[i] > max_dc_c)
				max_dc_c = cw_c[i];
		}

		fclose(fp);
	}
	catch(exception &e)
	{
		cout << "Error: " << e.what() << endl;
		destroy_code();

		exit(EXIT_FAILURE);
	}
}


void Ldpc_Code_cl::setup_layers(const char* clfile)
{
    FILE *fp = fopen(clfile, "r");
    if(!fp)
        throw runtime_error("Can not open layer file");

    fscanf(fp, "nl: %lu\n", &nl_c);

    lw_c = new uint64_t[nl_c];
    layers_c = new uint64_t*[nl_c];

    for (size_t i = 0; i < nl_c; ++i)
    {
        fscanf(fp, "cn[i]: %lu\n", &(lw_c[i]));
        layers_c[i] = new uint64_t[lw_c[i]];
        for (size_t j = 0; j < lw_c[i]; ++j)
            fscanf(fp, "%lu\n", &(layers_c[i][j]));
    }
    fclose(fp);
}


void Ldpc_Code_cl::setup_code_mgd(const char* filename)
{
	//init
    puncture_c = nullptr;
    shorten_c = nullptr;
    cw_c = nullptr;
    cn_c = nullptr;
    vw_c = nullptr;
    vn_c = nullptr;
    r_c = nullptr;
    c_c = nullptr;
    lw_c = nullptr;
    layers_c = nullptr;

    try
    {
        FILE *fp;

        fp = fopen(filename, "r");
        if(!fp)
            throw runtime_error("can not open codefile for reading.");

        fscanf(fp, "nc: %lu\n", &n_c);
        fscanf(fp, "mc: %lu\n", &m_c);
        fscanf(fp, "nct: %lu\n", &nct_c);
        fscanf(fp, "mct: %lu\n", &mct_c);
        fscanf(fp,  "nnz: %lu\n", &nnz_c);
        k_c = n_c-m_c;
        kct_c = nct_c-mct_c;

        fscanf(fp, "puncture [%lu]: ", &(num_puncture_c));
        num_puncture_sys_c = 0;
        num_puncture_par_c = 0;
        if(num_puncture_c != 0)
        {
			hipMallocManaged(&puncture_c, num_puncture_c*sizeof(size_t));
            for(size_t i = 0; i < num_puncture_c; i++)
            {
                fscanf(fp, " %lu ", &(puncture_c[i]));
                if(puncture_c[i] < k_c)
                    num_puncture_sys_c++;
                else
                    num_puncture_par_c++;
            }
        }

        fscanf(fp, "shorten [%lu]: ", &num_shorten_c);
        if(num_shorten_c != 0)
        {
			hipMallocManaged(&shorten_c, num_shorten_c*sizeof(size_t));
            for(size_t i = 0; i < num_shorten_c; i++)
                fscanf(fp, " %lu ", &(shorten_c[i]));
        }

        size_t* cw_tmp;
        size_t* vw_tmp;

		cw_tmp = new size_t[m_c]();
		vw_tmp = new size_t[n_c]();

		hipMallocManaged(&cw_c, m_c*sizeof(size_t));
		hipMallocManaged(&vw_c, n_c*sizeof(size_t));
		hipMallocManaged(&r_c, nnz_c*sizeof(size_t));
		hipMallocManaged(&c_c, nnz_c*sizeof(size_t));

		for(size_t i = 0; i < m_c; i++)
		{
	        cw_c[i] = 0;
	        cw_tmp[i] = 0;
	    }
	    for(size_t i = 0; i < n_c; i++)
		{
	        vw_c[i] = 0;
	        vw_tmp[i] = 0;
	    }
		
        for(size_t i = 0; i < nnz_c; i++)
        {
            fscanf(fp, "%lu %lu\n", &(r_c[i]), &(c_c[i]));
            cw_c[r_c[i]]++;
            vw_c[c_c[i]]++;
        }

		hipMallocManaged(&cn_c, m_c*sizeof(size_t*));
        for(size_t i = 0; i < m_c; i++)
			hipMallocManaged(&cn_c[i], cw_c[i]*sizeof(size_t));

		hipMallocManaged(&vn_c, n_c*sizeof(size_t*));
        for(size_t i = 0; i < n_c; i++)
			hipMallocManaged(&vn_c[i], vw_c[i]*sizeof(size_t));

        for(size_t i = 0; i < nnz_c; i++)
        {
            cn_c[r_c[i]][cw_tmp[r_c[i]]++] = i;
            vn_c[c_c[i]][vw_tmp[c_c[i]]++] = i;
        }

		delete[] cw_tmp;
		delete[] vw_tmp;

        max_dc_c = 0;
        for(size_t i = 0; i < m_c; i++)
        {
            if(cw_c[i] > max_dc_c)
                max_dc_c = cw_c[i];
        }

        fclose(fp);
    }
    catch(exception &e)
    {
        cout << "Error: " << e.what() << endl;
        destroy_code_mgd();

        exit(EXIT_FAILURE);
    }
}


void Ldpc_Code_cl::setup_layers_mgd(const char* clfile)
{
    FILE *fp = fopen(clfile, "r");
    if(!fp)
        throw runtime_error("Can not open layer file");

    fscanf(fp, "nl: %lu\n", &nl_c);

	hipMallocManaged(&lw_c, nl_c*sizeof(uint64_t));
	hipMallocManaged(&layers_c, nl_c*sizeof(uint64_t*));

    for (size_t i = 0; i < nl_c; ++i)
    {
        fscanf(fp, "cn[i]: %lu\n", &(lw_c[i]));
		hipMallocManaged(&layers_c[i], lw_c[i]*sizeof(uint64_t));
        for (size_t j = 0; j < lw_c[i]; ++j)
            fscanf(fp, "%lu\n", &(layers_c[i][j]));
    }
    fclose(fp);
}


void Ldpc_Code_cl::prefetch_code()
{
	hipDeviceSynchronize();

	int dev = -1;
	hipGetDevice(&dev);

	if(num_puncture_c != 0)	{ hipMemPrefetchAsync(puncture_c, sizeof(size_t)*num_puncture_c, dev, NULL); }
	if(num_shorten_c != 0) { hipMemPrefetchAsync(shorten_c, sizeof(size_t)*num_shorten_c, dev, NULL); }


	for(size_t i = 0; i < m_c; i++) {
		hipMemPrefetchAsync(cn_c[i], sizeof(size_t)*cw_c[i], dev, NULL);
	}
	hipMemPrefetchAsync(cn_c, sizeof(size_t*)*m_c, dev, NULL);


	for(size_t i = 0; i < m_c; i++) {
		hipMemPrefetchAsync(vn_c[i], sizeof(size_t)*vw_c[i], dev, NULL);
	}
	hipMemPrefetchAsync(vn_c, sizeof(size_t*)*n_c, dev, NULL);


	for (size_t i = 0; i < nl_c; ++i) {
		hipMemPrefetchAsync(layers_c[i], sizeof(uint64_t)*lw_c[i], dev, NULL);
	}
	hipMemPrefetchAsync(layers_c, sizeof(uint64_t*)*nl_c, dev, NULL);
	hipMemPrefetchAsync(lw_c, sizeof(uint64_t)*nl_c, dev, NULL);

	hipMemPrefetchAsync(cw_c, sizeof(size_t)*m_c, dev, NULL);
	hipMemPrefetchAsync(vw_c, sizeof(size_t)*n_c, dev, NULL);
	hipMemPrefetchAsync(r_c, sizeof(size_t)*nnz_c, dev, NULL);
	hipMemPrefetchAsync(c_c, sizeof(size_t)*nnz_c, dev, NULL);

	hipMemPrefetchAsync(this, sizeof(Ldpc_Code_cl), dev, NULL);
}


void Ldpc_Code_cl::destroy_code()
{
    if (vn_c != nullptr)
    {
        for(size_t i = 0; i < n_c; i++) { delete[] vn_c[i]; }
        delete[] vn_c;
    }

    if (vn_c != nullptr)
    {
        for(size_t i = 0; i < m_c; i++) { delete[] cn_c[i]; }
        delete[] cn_c;
    }

    if (vw_c != nullptr) { delete[] vw_c; }
    if (cw_c != nullptr) { delete[] cw_c; }
    if (r_c != nullptr) { delete[] r_c; }
    if (c_c != nullptr) { delete[] c_c; }
    if (puncture_c != nullptr) { delete[] puncture_c; }
    if (shorten_c != nullptr) { delete[] shorten_c; }
    if (layers_c != nullptr)
    {
        for(size_t i = 0; i < nl_c; i++) { delete[] layers_c[i]; }
        delete[] layers_c;
    }
    if (lw_c != nullptr) { delete[] lw_c; }
}


void Ldpc_Code_cl::destroy_code_mgd()
{
    if (vn_c != nullptr)
    {
        for(size_t i = 0; i < n_c; i++) { hipFree(vn_c[i]); }
        hipFree(vn_c);
    }
    if (vn_c != nullptr)
    {
        for(size_t i = 0; i < m_c; i++) { hipFree(cn_c[i]); }
        hipFree(cn_c);
    }
    if (vw_c != nullptr) { hipFree(vw_c); }
    if (cw_c != nullptr) { hipFree(cw_c); }
    if (r_c != nullptr) { hipFree(r_c); }
    if (c_c != nullptr) { hipFree(c_c); }
    if (puncture_c != nullptr) { hipFree(puncture_c); }
    if (shorten_c != nullptr) { hipFree(shorten_c); }
    if (layers_c != nullptr)
	{
        for(size_t i = 0; i < nl_c; i++) { hipFree(layers_c[i]); }
        hipFree(layers_c);
    }
    if (lw_c != nullptr) { hipFree(lw_c); }
}

void Ldpc_Code_cl::print_ldpc_code()
{
    cout << "=========== LDPC ===========" << endl;
    cout << "nc : " << n_c << endl;
    cout << "mc : " << m_c << endl;
    cout << "kc : " << k_c << endl;
    cout << "nnz : " << nnz_c << endl;
    cout << "nct :" << nct_c << endl;
    cout << "mct : " << mct_c << endl;
    cout << "kct : " << kct_c << endl;
    cout << "max dc : " << max_dc_c << endl;
    cout << "num puncture: " << num_puncture_c << endl;
    cout << "num puncture sys: " << num_puncture_sys_c << endl;
    cout << "num puncture par: " << num_puncture_par_c << endl;
    cout << "num shorten: " << num_shorten_c << endl;
    cout << "=========== LDPC: END ===========" << endl;

    printf("=========== LDPC LAYERS ===========\n");
    printf("nl: %lu\n", nl_c);
    for (size_t i = 0; i < nl_c; ++i)
    {
        printf("cn[%lu]: %lu\n", i, lw_c[i]);
        printVector<uint64_t>(layers_c[i], lw_c[i]);
        printf("\n");
    }
    printf("========= LDPC LAYERS: END ========\n");
}

__host__ __device__ uint64_t Ldpc_Code_cl::nc() const { return n_c; }
__host__ __device__ uint64_t Ldpc_Code_cl::kc() const { return k_c; }
__host__ __device__ uint64_t Ldpc_Code_cl::mc() const { return m_c; }
__host__ __device__ uint64_t Ldpc_Code_cl::nnz() const { return nnz_c; }
__host__ __device__ size_t *Ldpc_Code_cl::cw() const { return cw_c; }
__host__ __device__ size_t *Ldpc_Code_cl::vw() const { return vw_c; }
__host__ __device__ size_t **Ldpc_Code_cl::cn() const { return cn_c; }
__host__ __device__ size_t **Ldpc_Code_cl::vn() const { return vn_c; }
__host__ __device__ size_t *Ldpc_Code_cl::r() const { return r_c; }
__host__ __device__ size_t *Ldpc_Code_cl::c() const { return c_c; }
__host__ __device__ uint64_t Ldpc_Code_cl::nct() const { return nct_c; }
__host__ __device__ uint64_t Ldpc_Code_cl::mct() const { return mct_c; }
__host__ __device__ size_t *Ldpc_Code_cl::puncture() const { return puncture_c; }
__host__ __device__ size_t Ldpc_Code_cl::num_puncture() const { return num_puncture_c; }
__host__ __device__ size_t *Ldpc_Code_cl::shorten() const { return shorten_c; }
__host__ __device__ size_t Ldpc_Code_cl::num_shorten() const { return num_shorten_c; }
__host__ __device__ uint64_t Ldpc_Code_cl::kct() const { return kct_c; }
__host__ __device__ size_t Ldpc_Code_cl::max_dc() const { return max_dc_c; }
__host__ __device__ uint64_t Ldpc_Code_cl::nl() const { return nl_c; }
__host__ __device__ uint64_t *Ldpc_Code_cl::lw() const { return lw_c; }
__host__ __device__ uint64_t **Ldpc_Code_cl::layers() const { return layers_c; }


void ldpc::dec2bin(uint64_t val, uint8_t m)
{
    for(size_t i = 0; i < m; i++)
        printf("%lu", (val>>(m-i-1) & 0x01));
}

template<typename T> void ldpc::printVector(T *x, const size_t &l)
{
    cout << "[";
    for (size_t i = 0; i < l-1; ++i)
        cout << x[i] << " ";
    cout << x[l-1] << "]";
}

__host__ __device__ double ldpc::jacobian(const double& L1, const double& L2)
{
#ifdef CN_APPROX_LIN
    return sign(L1) * sign(L2) * fmin(fabs(L1),fabs(L2)) + jacobian_lin_approx(L1+L2) - jacobian_lin_approx(L1-L2);
#elif CN_APPROX_MINSUM
    return sign(L1) * sign(L2) * fmin(fabs(L1), fabs(L2));
#else
    return sign(L1) * sign(L2) * fmin(fabs(L1),fabs(L2)) + log((1+exp(-fabs(L1+L2)))/(1+exp(-fabs(L1-L2))));
#endif
}

__host__ __device__ double ldpc::jacobian_lin_approx(const double& L)
{
    double Labs = fabs(L);

    if(Labs < 1.0) {
        return -0.375 * Labs  + 0.6825;
    } else if((Labs >= 1.0) && (Labs < 2.625)) {
        return -0.1875 * Labs + 0.5;
    } else {
        return 0;
    }
}

__host__ __device__ int8_t ldpc::sign(const double& a)
{
    return (a <= 0) ? -1 : 1;
}
