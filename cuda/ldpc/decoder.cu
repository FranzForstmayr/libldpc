#include "hip/hip_runtime.h"
#include "ldpc.cuh"
#include <exception>

using namespace ldpc;
using namespace std;


Ldpc_Decoder_cl::Ldpc_Decoder_cl(Ldpc_Code_cl* code, const uint16_t I, const bool early_term, const bool mgd)
: ldpc_code(code), max_iter(I), early_termination(early_term)
{
    if (mgd)
    {
        setup_dec_mgd();
        prefetch_dec();
    }
    else
    {
        //setup_dec();
    }

    isMgd = mgd;
}


Ldpc_Decoder_cl::~Ldpc_Decoder_cl()
{
    if (isMgd)
    {
        destroy_dec_mgd();
    }
    else
    {
        //destroy_dec();
    }
}


void Ldpc_Decoder_cl::setup_dec_mgd()
{
    block_size = 256;
    num_blocks = ceil((ldpc_code->nnz() + block_size - 1) / block_size);

    l_c2v = nullptr;
    l_v2c = nullptr;
    f = nullptr;
    b = nullptr;
    fb_ref = nullptr;
    lsum = nullptr;
    l_c2v_pre = nullptr;
    c_out = nullptr;
    synd = nullptr;
    llr_in = nullptr;
    llr_out = nullptr;

    const uint64_t num_layers = ldpc_code->nl();

    try
    {
        //num layers times num nnz
        hipMallocManaged(&l_c2v, sizeof(double)*num_layers*ldpc_code->nnz());
        if (l_c2v == NULL || l_c2v == nullptr) { throw runtime_error("l_c2v alloc failed."); }
        hipMallocManaged(&l_v2c, sizeof(double)*num_layers*ldpc_code->nnz());
        if (l_v2c == NULL || l_v2c == nullptr) { throw runtime_error("l_v2c alloc failed."); }
        hipMallocManaged(&l_c2v_pre, sizeof(double)*num_layers*ldpc_code->nnz());
        if (l_c2v_pre == NULL || l_c2v_pre == nullptr) { throw runtime_error("l_c2v_pre alloc failed."); }

        hipMallocManaged(&f, sizeof(double)*num_layers*ldpc_code->max_dc());
        if (f == NULL || f == nullptr) { throw runtime_error("f alloc failed."); }
        hipMallocManaged(&b, sizeof(double)*num_layers*ldpc_code->max_dc());
        if (b == NULL || b == nullptr) { throw runtime_error("b alloc failed."); }

        hipMallocManaged(&fb_ref, ldpc_code->max_dc());
        if (fb_ref == NULL || fb_ref == nullptr) { throw runtime_error("fb_ref alloc failed."); }

        hipMallocManaged(&lsum, sizeof(double)*ldpc_code->nnz());
        if (lsum == NULL || lsum == nullptr) { throw runtime_error("lsum alloc failed."); }

        hipMallocManaged(&llr_in, sizeof(double)*ldpc_code->nc());
        if (llr_in == NULL || llr_in == nullptr) { throw runtime_error("llr_in alloc failed."); }
        hipMallocManaged(&llr_out, sizeof(double)*ldpc_code->nc());
        if (llr_out == NULL || llr_out == nullptr) { throw runtime_error("llr_out alloc failed."); }
        hipMallocManaged(&c_out, sizeof(bits_t)*ldpc_code->nc());
        if (c_out == NULL || c_out == nullptr) { throw runtime_error("c_out alloc failed."); }

        hipMallocManaged(&synd, sizeof(bits_t)*ldpc_code->mc());
        if (synd == NULL || synd == nullptr) { throw runtime_error("synd alloc failed."); }
    }
    catch (exception& e)
    {
        cout << "Error: " << e.what() << endl;
        destroy_dec_mgd();
        exit(EXIT_FAILURE);
    }
}

void Ldpc_Decoder_cl::prefetch_dec()
{
    hipDeviceSynchronize();

    int dev = -1;
    hipGetDevice(&dev);

    const uint64_t num_layers = ldpc_code->nl();

    hipMemPrefetchAsync(l_c2v, sizeof(double)*num_layers*ldpc_code->nnz(), dev, NULL);
    hipMemPrefetchAsync(l_v2c, sizeof(double)*num_layers*ldpc_code->nnz(), dev, NULL);
    hipMemPrefetchAsync(l_c2v_pre, sizeof(double)*num_layers*ldpc_code->nnz(), dev, NULL);

    hipMemPrefetchAsync(fb_ref, ldpc_code->max_dc(), dev, NULL);

    hipMemPrefetchAsync(lsum, sizeof(double)*ldpc_code->nnz(), dev, NULL);

    hipMemPrefetchAsync(llr_in, sizeof(double)*ldpc_code->nc(), dev, NULL);
    hipMemPrefetchAsync(llr_out, sizeof(double)*ldpc_code->nc(), dev, NULL);
    hipMemPrefetchAsync(c_out, sizeof(double)*ldpc_code->nc(), dev, NULL);

    hipMemPrefetchAsync(synd, sizeof(double)*ldpc_code->mc(), dev, NULL);

    hipMemPrefetchAsync(this, sizeof(Ldpc_Decoder_cl), dev, NULL);
}

void Ldpc_Decoder_cl::destroy_dec_mgd()
{
    if (l_c2v != nullptr) { hipFree(l_c2v); }
    if (l_v2c != nullptr) { hipFree(l_v2c); }
    if (l_c2v_pre != nullptr) { hipFree(l_c2v_pre); }
    if (f != nullptr) { hipFree(f); }
    if (b != nullptr) { hipFree(b); }
    if (fb_ref != nullptr) { hipFree(fb_ref); }
    if (lsum != nullptr) { hipFree(lsum); }
    if (c_out != nullptr) { hipFree(c_out); }
    if (synd != nullptr) { hipFree(synd); }
    if (llr_in != nullptr) { hipFree(llr_in); }
    if (llr_out != nullptr) { hipFree(llr_out); }
}


__host__ __device__ bool Ldpc_Decoder_cl::is_codeword()
{
    is_cw = true;

    //calc syndrome
    cudakernel::decoder::calc_synd<<<get_num_size(ldpc_code->mc(), 256), 256>>>(this);
    hipDeviceSynchronize();

    return is_cw;
}


__host__ __device__ bool Ldpc_Decoder_cl::is_codeword_legacy()
{
    bool is_codeword = true;

    //calc syndrome
    bits_t s;
    for (size_t i = 0; i < ldpc_code->mc(); i++)
    {
        s = 0;
        for (size_t j = 0; j < ldpc_code->cw()[i]; j++)
            s ^= c_out[ldpc_code->c()[ldpc_code->cn()[i][j]]];

        if (s)
        {
            return false;
        }
    }

    return is_codeword;
}


uint16_t Ldpc_Decoder_cl::decode_legacy()
{
    uint16_t it;

    size_t* vn;
    size_t* cn;

    size_t vw;
    size_t cw;

    //initialize with llrs
    for(size_t i = 0; i < ldpc_code->nnz(); i++) {
        l_v2c[i] = llr_in[ldpc_code->c()[i]];
    }

    it = 0;
    while(it < max_iter) {
        for(size_t i = 0; i < ldpc_code->mc(); i++) {
            cw = ldpc_code->cw()[i];
            cn = ldpc_code->cn()[i];
            f[0] = l_v2c[*cn];
            b[cw-1] = l_v2c[*(cn+cw-1)];
            for(size_t j = 1; j < cw; j++) {
                f[j] = jacobian(f[j-1], l_v2c[*(cn+j)]);
                b[cw-1-j] = jacobian(b[cw-j], l_v2c[*(cn + cw-j-1)]);
            }

            l_c2v[*cn] = b[1];
            l_c2v[*(cn+cw-1)] = f[cw-2];
            for(size_t j = 1; j < cw-1; j++) {
                l_c2v[*(cn+j)] = jacobian(f[j-1], b[j+1]);
            }
        }

        // VN node processing
        for(size_t i = 0; i < ldpc_code->nc(); i++) {
            double tmp = llr_in[i];
            vw = ldpc_code->vw()[i];
            vn = ldpc_code->vn()[i];
            while(vw--) {
                tmp += l_c2v[*vn++];
            }
            vn = ldpc_code->vn()[i];
            vw = ldpc_code->vw()[i];
            while(vw--) {
                l_v2c[*vn] = tmp - l_c2v[*vn];
                vn++;
            }
        }

        // app calculation
        for(size_t i = 0; i < ldpc_code->nc(); i++) {
            llr_out[i] = llr_in[i];
            vn = ldpc_code->vn()[i];
            vw = ldpc_code->vw()[i];
            while(vw--) {
                llr_out[i] += l_c2v[*vn++];
            }
            c_out[i] = (llr_out[i] <= 0);
        }

        it++;

        if (early_termination) {
            if (is_codeword_legacy()) {
                break;
            }
        }
    }

    return it;
}


uint16_t Ldpc_Decoder_cl::decode_layered_legacy()
{
    size_t* vn;
    size_t* cn;

    size_t vw;
    size_t cw;

    size_t i_nnz;
    size_t i_dc;

    //initialize
    for (size_t i = 0; i < ldpc_code->nnz(); ++i)
    {
        lsum[i] = 0.0;
        for (size_t l = 0; l < ldpc_code->nl(); ++l)
        {
            l_c2v[l*ldpc_code->nnz()+i] = 0.0;
            l_v2c[l*ldpc_code->nnz()+i] = 0.0;
            l_c2v_pre[l*ldpc_code->nnz()+i] = 0.0;
        }
    }

    uint16_t I = 0;
    while (I < max_iter)
    {
        for (size_t l = 0; l < ldpc_code->nl(); ++l)
        {
            i_nnz = l*ldpc_code->nnz();
            i_dc = l*ldpc_code->max_dc();

            // VN node intialization
            for(size_t i = 0; i < ldpc_code->nc(); i++)
            {
                double tmp = llr_in[i];
                vw = ldpc_code->vw()[i];
                vn = ldpc_code->vn()[i];
                while(vw--)
                    tmp += lsum[*vn++];

                vn = ldpc_code->vn()[i];
                vw = ldpc_code->vw()[i];
                while(vw--)
                {
                    l_v2c[i_nnz + *vn] = tmp - l_c2v[i_nnz + *vn];
                    ++vn;
                }
            }

            //CN processing
            for(size_t i = 0; i < ldpc_code->lw()[l]; i++)
            {
                cw = ldpc_code->cw()[ldpc_code->layers()[l][i]];
                cn = ldpc_code->cn()[ldpc_code->layers()[l][i]];
                f[i_dc] = l_v2c[i_nnz + *cn];
                b[i_dc + cw-1] = l_v2c[i_nnz + *(cn+cw-1)];
                for(size_t j = 1; j < cw; j++)
                {
                    f[i_dc + j] = jacobian(f[i_dc + j-1], l_v2c[i_nnz + *(cn+j)]);
                    b[i_dc + cw-1-j] = jacobian(b[i_dc + cw-j], l_v2c[i_nnz + *(cn + cw-j-1)]);
                }

                l_c2v[i_nnz + *cn] = b[i_dc + 1];
                l_c2v[i_nnz + *(cn+cw-1)] = f[i_dc + cw-2];

                for(size_t j = 1; j < cw-1; j++)
                    l_c2v[i_nnz + *(cn+j)] = jacobian(f[i_dc + j-1], b[i_dc + j+1]);
            }

            //update the llr sum of layers, by replacing old llr of lyr l with new value
            for (size_t i = 0; i < ldpc_code->nnz(); ++i)
            {
                lsum[i] += l_c2v[i_nnz + i] - l_c2v_pre[i_nnz + i];
                l_c2v_pre[i_nnz + i] = l_c2v[i_nnz + i];
            }

            // app calculation
            for(size_t i = 0; i < ldpc_code->nc(); ++i)
            {
                llr_out[i] = llr_in[i];
                vn = ldpc_code->vn()[i];
                vw = ldpc_code->vw()[i];
                while(vw--)
                    llr_out[i] += lsum[*vn++];
                c_out[i] = (llr_out[i] <= 0);
            }

            if (early_termination)
            {
                if (is_codeword_legacy())
                {
                    return I;
                }
            }
        }

        ++I;
    }

    return I;
}


uint16_t Ldpc_Decoder_cl::decode_layered()
{
    cudakernel::decoder::decode_layered<<<1, 1>>>(this);
    hipDeviceSynchronize();

    return iter;
}


//tmpl fcts need definition in each file?
template<typename T> void ldpc::printVector(T *x, const size_t &l)
{
    cout << "[";
    for (size_t i = 0; i < l-1; ++i)
        cout << x[i] << " ";
    cout << x[l-1] << "]";
}


/*
 *	Cudakernels
 */
__global__ void cudakernel::decoder::clean_decoder(Ldpc_Decoder_cl* dec_mgd)
{
    uint_fast32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    uint_fast32_t stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < dec_mgd->ldpc_code->nnz(); i += stride)
    {
        dec_mgd->lsum[i] = 0.0;
        for (size_t l = 0; l < dec_mgd->ldpc_code->nl(); ++l)
        {
            dec_mgd->l_c2v[l*dec_mgd->ldpc_code->nnz()+i] = 0.0;
            dec_mgd->l_v2c[l*dec_mgd->ldpc_code->nnz()+i] = 0.0;
            dec_mgd->l_c2v_pre[l*dec_mgd->ldpc_code->nnz()+i] = 0.0;
        }
    }
}


__global__ void cudakernel::decoder::decode_layered(Ldpc_Decoder_cl* dec_mgd)
{
    size_t i_nnz;

    //zero everything out
    cudakernel::decoder::clean_decoder<<<get_num_size(dec_mgd->ldpc_code->nnz(), 256), 256>>>(dec_mgd);

    uint16_t I = 0;
    while (I < dec_mgd->max_iter)
    {
        for (uint64_t l = 0; l < dec_mgd->ldpc_code->nl(); ++l)
        {
            i_nnz = dec_mgd->ldpc_code->nnz()*l;

            //launch kernels here
            cudakernel::decoder::decode_lyr_vnupdate<<<get_num_size(dec_mgd->ldpc_code->nc(), 256), 256>>>(dec_mgd, i_nnz);
            cudakernel::decoder::decode_lyr_cnupdate<<<get_num_size(dec_mgd->ldpc_code->lw()[l], 256), 256>>>(dec_mgd, i_nnz, l);
            cudakernel::decoder::decode_lyr_sumllr<<<get_num_size(dec_mgd->ldpc_code->nnz(), 256), 256>>>(dec_mgd, i_nnz);
            cudakernel::decoder::decode_lyr_appcalc<<<get_num_size(dec_mgd->ldpc_code->nc(), 256), 256>>>(dec_mgd);

            if (dec_mgd->early_termination)
            {
                if (dec_mgd->is_codeword_legacy()) //break
                {
                    //l = dec_mgd->ldpc_code->nl();
                    //I += dec_mgd->max_iter;
                    goto break_here;
                }
            }
        }

        ++I;
    }

    break_here:
    hipDeviceSynchronize();

    dec_mgd->iter = I;
}


__global__ void cudakernel::decoder::decode_lyr_vnupdate(Ldpc_Decoder_cl* dec_mgd, size_t i_nnz)
{
    size_t* vn;
    size_t vw;

    uint_fast32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    uint_fast32_t stride = blockDim.x * gridDim.x;

    //VN processing
    for (size_t i = index; i < dec_mgd->ldpc_code->nc(); i += stride)
    {
        double tmp = dec_mgd->llr_in[i];
        vw =  dec_mgd->ldpc_code->vw()[i];
        vn = dec_mgd->ldpc_code->vn()[i];
        while(vw--)
            tmp += dec_mgd->lsum[*vn++];

        vn = dec_mgd->ldpc_code->vn()[i];
        vw = dec_mgd->ldpc_code->vw()[i];
        while(vw--)
        {
            dec_mgd->l_v2c[i_nnz + *vn] = tmp - dec_mgd->l_c2v[i_nnz + *vn];
            ++vn;
        }
    }
}


__global__ void cudakernel::decoder::decode_lyr_cnupdate(Ldpc_Decoder_cl* dec_mgd, size_t i_nnz, uint64_t l)
{
    size_t* cn;
    size_t cw;

    uint_fast32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    uint_fast32_t stride = blockDim.x * gridDim.x;

    double f_tmp[sizeof(dec_mgd->fb_ref)];
    double b_tmp[sizeof(dec_mgd->fb_ref)];

    //CN processing
    for (size_t i = index; i < dec_mgd->ldpc_code->lw()[l]; i += stride)
    {
        cw = dec_mgd->ldpc_code->cw()[dec_mgd->ldpc_code->layers()[l][i]];
        cn = dec_mgd->ldpc_code->cn()[dec_mgd->ldpc_code->layers()[l][i]];
        f_tmp[0] = dec_mgd->l_v2c[i_nnz + *cn];
        b_tmp[cw-1] = dec_mgd->l_v2c[i_nnz + *(cn+cw-1)];
        for(size_t j = 1; j < cw; j++)
        {
            f_tmp[j] = jacobian(f_tmp[j-1], dec_mgd->l_v2c[i_nnz + *(cn+j)]);
            b_tmp[cw-1-j] = jacobian(b_tmp[cw-j], dec_mgd->l_v2c[i_nnz + *(cn + cw-j-1)]);
        }

        dec_mgd->l_c2v[i_nnz + *cn] = b_tmp[1];
        dec_mgd->l_c2v[i_nnz + *(cn+cw-1)] = f_tmp[cw-2];

        for(size_t j = 1; j < cw-1; j++)
            dec_mgd->l_c2v[i_nnz + *(cn+j)] = jacobian(f_tmp[j-1], b_tmp[j+1]);
    }
}


__global__ void cudakernel::decoder::decode_lyr_sumllr(Ldpc_Decoder_cl* dec_mgd, size_t i_nnz)
{
    uint_fast32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    uint_fast32_t stride = blockDim.x * gridDim.x;

    //sum llrs
    for (size_t i = index; i < dec_mgd->ldpc_code->nnz(); i += stride)
    {
        dec_mgd->lsum[i] += dec_mgd->l_c2v[i_nnz + i] - dec_mgd->l_c2v_pre[i_nnz + i];
        dec_mgd->l_c2v_pre[i_nnz + i] = dec_mgd->l_c2v[i_nnz + i];
    }
}


__global__ void cudakernel::decoder::decode_lyr_appcalc(Ldpc_Decoder_cl* dec_mgd)
{
    size_t* vn;
    size_t vw;

    uint_fast32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    uint_fast32_t stride = blockDim.x * gridDim.x;

    //app calc
    for (size_t i = index; i < dec_mgd->ldpc_code->nc(); i += stride)
    {
        dec_mgd->llr_out[i] = dec_mgd->llr_in[i];
        vn = dec_mgd->ldpc_code->vn()[i];
        vw = dec_mgd->ldpc_code->vw()[i];
        while(vw--)
            dec_mgd->llr_out[i] += dec_mgd->lsum[*vn++];
        dec_mgd->c_out[i] = (dec_mgd->llr_out[i] <= 0);
    }
}


__global__ void cudakernel::decoder::calc_synd(Ldpc_Decoder_cl* dec_mgd)
{
    uint_fast32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    uint_fast32_t stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < dec_mgd->ldpc_code->mc(); i += stride)
    {
        dec_mgd->synd[i] = 0;
        for (size_t j = 0; j < dec_mgd->ldpc_code->cw()[i]; j++)
        {
            dec_mgd->synd[i] ^= dec_mgd->c_out[dec_mgd->ldpc_code->c()[dec_mgd->ldpc_code->cn()[i][j]]];
        }

        if (dec_mgd->synd[i])
        {
            dec_mgd->is_cw = false;
        }
    }
}
