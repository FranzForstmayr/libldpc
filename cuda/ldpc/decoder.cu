#include "hip/hip_runtime.h"
#include "ldpc.cuh"
#include <exception>

using namespace ldpc;
using namespace std;


__host__ __device__ Ldpc_Decoder_cl::Ldpc_Decoder_cl() {}
Ldpc_Decoder_cl::Ldpc_Decoder_cl(Ldpc_Code_cl* code) { setup_decoder(code); }
__host__ __device__ Ldpc_Decoder_cl::~Ldpc_Decoder_cl()
{
	if (init)
		destroy_dec();
}

void Ldpc_Decoder_cl::setup_decoder(Ldpc_Code_cl* code)
{
	init = true;
	ldpc_code = code;

	l_c2v = nullptr;
	l_v2c = nullptr;
	f = nullptr;
	b = nullptr;
	lsum = nullptr;

	c_out = nullptr;
	synd = nullptr;

	#ifdef QC_LYR_DEC
	const uint64_t num_layers = ldpc_code->nl();
	#else
	const uint64_t num_layers = 1;
	#endif

	try
	{
		//num layers times num nnz
		l_c2v = new double[num_layers * ldpc_code->nnz()]();
		l_v2c = new double[num_layers * ldpc_code->nnz()]();
		f = new double[num_layers * ldpc_code->max_dc()]();
		b = new double[num_layers * ldpc_code->max_dc()]();

		lsum = new double[ldpc_code->nnz()]();

		c_out = new bits_t[ldpc_code->nc()]();
		synd = new bits_t[ldpc_code->nc()]();
	}
	catch (exception& e)
	{
		cout << "Error: " << e.what() << endl;
		destroy_dec();
		exit(EXIT_FAILURE);
	}
}

__device__ void Ldpc_Decoder_cl::setup_decoder_device(Ldpc_Code_cl* code)
{
	init = true;
	ldpc_code = code;

	l_c2v = nullptr;
	l_v2c = nullptr;
	f = nullptr;
	b = nullptr;
	lsum = nullptr;

	c_out = nullptr;
	synd = nullptr;

	const uint64_t num_layers = ldpc_code->nl();

	//num layers times num nnz
	l_c2v = new double[num_layers * ldpc_code->nnz()]();
	l_v2c = new double[num_layers * ldpc_code->nnz()]();
	f = new double[num_layers * ldpc_code->max_dc()]();
	b = new double[num_layers * ldpc_code->max_dc()]();

	lsum = new double[ldpc_code->nnz()]();

	c_out = new bits_t[ldpc_code->nc()]();
	synd = new bits_t[ldpc_code->nc()]();
}

__host__ __device__ void Ldpc_Decoder_cl::destroy_dec()
{
    if (l_c2v != nullptr)
        delete[] l_c2v;
    if (l_v2c != nullptr)
        delete[] l_v2c;
    if (f != nullptr)
        delete[] f;
    if (b != nullptr)
        delete[] b;
    if (lsum != nullptr)
        delete[] lsum;
    if (c_out != nullptr)
        delete[] c_out;
    if (synd != nullptr)
        delete[] synd;
}


__host__ __device__ bool Ldpc_Decoder_cl::is_codeword()
{
    bool is_codeword = true;

    //calc syndrome
    bits_t s;
    for (size_t i = 0; i < ldpc_code->mc(); i++)
    {
        s = 0;
        for (size_t j = 0; j < ldpc_code->cw()[i]; j++)
            s ^= c_out[ldpc_code->c()[ldpc_code->cn()[i][j]]];

        if (s)
        {
            return false;
        }
    }

    return is_codeword;
}

uint64_t Ldpc_Decoder_cl::decode_legacy(double* llr_in, double* llr_out, const uint64_t& max_iter, const bool& early_termination)
{
    size_t it;

    size_t* vn;
    size_t* cn;

    size_t vw;
    size_t cw;

    /* initialize with llrs */
    for(size_t i = 0; i < ldpc_code->nnz(); i++) {
        l_v2c[i] = llr_in[ldpc_code->c()[i]];
    }

    it = 0;
    while(it < max_iter) {
        for(size_t i = 0; i < ldpc_code->mc(); i++) {
            cw = ldpc_code->cw()[i];
            cn = ldpc_code->cn()[i];
            f[0] = l_v2c[*cn];
            b[cw-1] = l_v2c[*(cn+cw-1)];
            for(size_t j = 1; j < cw; j++) {
                f[j] = jacobian(f[j-1], l_v2c[*(cn+j)]);
                b[cw-1-j] = jacobian(b[cw-j], l_v2c[*(cn + cw-j-1)]);
            }

            l_c2v[*cn] = b[1];
            l_c2v[*(cn+cw-1)] = f[cw-2];
            for(size_t j = 1; j < cw-1; j++) {
                l_c2v[*(cn+j)] = jacobian(f[j-1], b[j+1]);
            }
        }

        /* VN node processing */
        for(size_t i = 0; i < ldpc_code->nc(); i++) {
            double tmp = llr_in[i];
            vw = ldpc_code->vw()[i];
            vn = ldpc_code->vn()[i];
            while(vw--) {
                tmp += l_c2v[*vn++];
            }
            vn = ldpc_code->vn()[i];
            vw = ldpc_code->vw()[i];
            while(vw--) {
                l_v2c[*vn] = tmp - l_c2v[*vn];
                vn++;
            }
        }

        // app calculation
        for(size_t i = 0; i < ldpc_code->nc(); i++) {
            llr_out[i] = llr_in[i];
            vn = ldpc_code->vn()[i];
            vw = ldpc_code->vw()[i];
            while(vw--) {
                llr_out[i] += l_c2v[*vn++];
            }
            c_out[i] = (llr_out[i] <= 0);
        }

        it++;

        if (early_termination) {
            if (is_codeword()) {
                break;
            }
        }
    }

    return it;
}

__global__ void cudakernel::setup_decoder(Ldpc_Code_cl* code_managed, Ldpc_Decoder_cl** dec_ptr)
{
	*dec_ptr = new Ldpc_Decoder_cl();
	(**dec_ptr).setup_decoder_device(code_managed);
	printf("Cuda Device :: Decoder set up!\n");
}

__global__ void cudakernel::destroy_decoder(Ldpc_Decoder_cl** dec_ptr)
{
	delete *dec_ptr;
	printf("Cuda Device :: Decoder destroyed!\n");
}



//tmpl fcts need definition in each file?
template<typename T> void ldpc::printVector(T *x, const size_t &l)
{
    cout << "[";
    for (size_t i = 0; i < l-1; ++i)
        cout << x[i] << " ";
    cout << x[l-1] << "]";
}
