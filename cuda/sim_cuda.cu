#include "ldpc/ldpc.cuh"
#include "simulation.cuh"

using namespace ldpc;
using namespace std;


//nvcc -std=c++11 sim_cuda.cu simulation.cu ldpc/ldpc.cu ldpc/decoder.cu -o sim_cuda
// ??? -arch sm_35 -rdc=true
int main()
{
	Ldpc_Code_cl* code_managed;// = new Ldpc_Code_cl("../src/code/test_code/code_rand_proto_3x6_400_4.txt", "../src/code/test_code/layer_rand_proto_3x6_400_4.txt");
	hipMallocManaged(&code_managed, sizeof(Ldpc_Code_cl));
	code_managed->setup_code_managed("../src/code/test_code/code_rand_proto_3x6_400_4.txt", "../src/code/test_code/layer_rand_proto_3x6_400_4.txt");

	Ldpc_Decoder_cl** dec_ptr;
	hipMallocManaged(&dec_ptr, sizeof(Ldpc_Decoder_cl*));

	double *llrin, *llrout;
	hipMallocManaged(&llrin, code_managed->nc()*sizeof(double));
	hipMallocManaged(&llrout, code_managed->nc()*sizeof(double));
	for (size_t i=0; i<code_managed->nc(); ++i)
	{
		llrin[i] = Sim_AWGN_cl::randn();
		llrout[i] = 0.0;
	}


	cudakernel::setup_decoder<<<1, 1>>>(code_managed, dec_ptr);
	cudakernel::decode<<<1, 1>>>(dec_ptr, llrin, llrout, 1, false);

	hipDeviceSynchronize();

	//printVector<double>(llrout, code_managed->nc());


	cudakernel::destroy_decoder<<<1, 1>>>(dec_ptr);

	hipFree(llrin);
	hipFree(llrout);
	code_managed->destroy_ldpc_code_managed();
	hipFree(code_managed);
	hipFree(dec_ptr);

	return 0;
}


//tmpl fcts need definition in each file?
template<typename T> void ldpc::printVector(T *x, const size_t &l)
{
	cout << "[";
	for (size_t i = 0; i < l-1; ++i)
	cout << x[i] << " ";
	cout << x[l-1] << "]";
}
