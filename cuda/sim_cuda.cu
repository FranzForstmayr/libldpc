#include "ldpc/ldpc.cuh"
#include "simulation.cuh"

using namespace ldpc;
using namespace std;


//nvcc -std=c++11 sim_cuda.cu simulation.cu ldpc/ldpc.cu ldpc/decoder.cu -o sim_cuda -arch sm_35 -rdc=true -O3
int main()
{
	Ldpc_Code_cl* code_managed;
	hipMallocManaged(&code_managed, sizeof(Ldpc_Code_cl));
	*code_managed = Ldpc_Code_cl();
	code_managed->setup_code_managed("../src/code/test_code/code10K.txt", "../src/code/test_code/layer10K.txt");

	Sim_AWGN_cl sim = Sim_AWGN_cl(code_managed, "../src/sim.txt", "../src/code/test_code/map10K.txt");

	Ldpc_Decoder_cl** dec_ptr;
	hipMallocManaged(&dec_ptr, sizeof(Ldpc_Decoder_cl*));

	double *llrin, *llrout;
	hipMallocManaged(&llrin, code_managed->nc()*sizeof(double));
	hipMallocManaged(&llrout, code_managed->nc()*sizeof(double));
	for (size_t i=0; i<code_managed->nc(); ++i)
	{
		llrin[i] = Sim_AWGN_cl::randn();
		llrout[i] = 0.0;
	}


	cudakernel::setup_decoder<<<1, 1>>>(code_managed, dec_ptr);


	TIME_PROF("GPU", sim.decode_lyr(dec_ptr, llrin, llrout, 50, false), "ms");

	Ldpc_Decoder_cl dec = Ldpc_Decoder_cl(code_managed);
	TIME_PROF("CPU", dec.decode_layered_legacy(llrin, llrout, 50, false), "ms");

	cudakernel::destroy_decoder<<<1, 1>>>(dec_ptr);

	hipFree(llrin);
	hipFree(llrout);
	code_managed->destroy_ldpc_code_managed();
	hipFree(code_managed);
	hipFree(dec_ptr);

	return 0;
}


//tmpl fcts need definition in each file?
template<typename T> void ldpc::printVector(T *x, const size_t &l)
{
	cout << "[";
	for (size_t i = 0; i < l-1; ++i)
	cout << x[i] << " ";
	cout << x[l-1] << "]";
}

/*
struct timespec tstart={0,0}, tend={0,0};
clock_gettime(CLOCK_MONOTONIC, &tstart);

clock_gettime(CLOCK_MONOTONIC, &tend);
printf("Time GPU: %.5f ms\n", (((double)tend.tv_nsec) - ((double)tstart.tv_nsec))*1e-6);
*/
