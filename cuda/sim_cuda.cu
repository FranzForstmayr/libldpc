#include "ldpc/ldpc.cuh"
#include "simulation.cuh"

using namespace ldpc;
using namespace std;


//nvcc -std=c++11 sim_cuda.cu simulation.cu ldpc/ldpc.cu ldpc/decoder.cu -o sim_cuda -arch sm_35 -rdc=true -O3
int main()
{
	//set up code class on unified memory 
	Ldpc_Code_cl* code_managed;
	hipMallocManaged(&code_managed, sizeof(Ldpc_Code_cl));
	*code_managed = Ldpc_Code_cl();
	code_managed->setup_code_managed("../src/code/test_code/code_rand_proto_3x6_400_4.txt", "../src/code/test_code/layer_rand_proto_3x6_400_4.txt");

	//set up simulation
	//Sim_AWGN_cl sim = Sim_AWGN_cl(code_managed, "../src/sim.txt", "../src/code/test_code/map10K.txt");

	double *llrin, *llrout;
	hipMallocManaged(&llrin, code_managed->nc()*sizeof(double));
	hipMallocManaged(&llrout, code_managed->nc()*sizeof(double));
	for (size_t i=0; i<code_managed->nc(); ++i)
	{
		llrin[i] = Sim_AWGN_cl::randn();
		llrout[i] = 0.0;
	}

	//set up decoder on unified memory
	Ldpc_Decoder_cl* dec_ufd;
	hipMallocManaged(&dec_ufd, sizeof(Ldpc_Decoder_cl));
	dec_ufd->setup_decoder_managed(code_managed);

	for (int i=0; i<100; ++i)
		TIME_PROF("GPU", dec_ufd->decode_layered(llrin, llrout, 50, false), "ms");

	TIME_PROF("CPU", dec_ufd->decode_layered_legacy(llrin, llrout, 50, false), "ms");


	hipFree(llrin);
	hipFree(llrout);
	
	//destroy decoder
	dec_ufd->destroy_dec_managed();
	hipFree(dec_ufd);
	
	//destroy code
	code_managed->destroy_ldpc_code_managed();
	hipFree(code_managed);

	return 0;
}


//tmpl fcts need definition in each file?
template<typename T> void ldpc::printVector(T *x, const size_t &l)
{
	cout << "[";
	for (size_t i = 0; i < l-1; ++i)
	cout << x[i] << " ";
	cout << x[l-1] << "]";
}
